#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <fstream>
#include <iostream>
#include <limits>
#include <string>
#include <algorithm>

#include "SimulationNBodyCudaAoS.cuh"

#define MAX_SHARED_PER_BLOCK 48000
#define THREADS_PER_BLK 512

namespace cuda
{
  void printGPUInfo()
  {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    printf("Number of devices: %d\n", nDevices);

    for (int i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (MHz): %d\n",
               prop.memoryClockRate / 1024);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
               2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        printf("  Total global memory (Gbytes) %.1f\n", (float)(prop.totalGlobalMem) / 1024.0 / 1024.0 / 1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n", (float)(prop.sharedMemPerBlock) / 1024.0);
        printf("  minor-major: %d-%d\n", prop.minor, prop.major);
        printf("  Warp-size: %d\n", prop.warpSize);
        printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n", prop.deviceOverlap ? "yes" : "no");
    }
  }

  __global__ void computeBodiesAccellAoS_k(float4 *d_AoS, float3 *d_acc, const unsigned long nBodies, const float softSquared, const float G)
  {
    __shared__ float4 shared_mem[THREADS_PER_BLK];

    const unsigned long iBody = blockIdx.x * blockDim.x + threadIdx.x;
    float4 myBody;
    if(iBody < nBodies)
    {
      myBody = d_AoS[iBody];
    }
    float3 acc = {0.f, 0.f, 0.f};

    unsigned tileIdx;
    unsigned tile; 
    for(tile = 0; tile < nBodies / THREADS_PER_BLK; tile++)
    {
      tileIdx = tile * THREADS_PER_BLK + threadIdx.x;
      shared_mem[threadIdx.x] = d_AoS[tileIdx];
      __syncthreads();
      #pragma unroll 4
      for(unsigned jBody = 0; jBody < THREADS_PER_BLK; jBody++)
      {
        float4 otherBody = shared_mem[jBody];
        float3 rij = {otherBody.x - myBody.x, otherBody.y - myBody.y, otherBody.z - myBody.z};
        float rijSquared = rij.x * rij.x + rij.y * rij.y + rij.z * rij.z + softSquared;

        float ai = G * otherBody.w / (rijSquared * sqrtf(rijSquared));

        acc.x += ai * rij.x;
        acc.y += ai * rij.y;
        acc.z += ai * rij.z;
      }
      __syncthreads();
    }

    // compute epilogue
    tileIdx = tile * THREADS_PER_BLK + threadIdx.x;
    //load the last tile
    shared_mem[threadIdx.x] = (tileIdx < nBodies) ? d_AoS[tileIdx] : make_float4(0.f, 0.f, 0.f, 0.f);
    __syncthreads();

    for(unsigned jBody = 0; jBody < nBodies % THREADS_PER_BLK; jBody++)
    {
      float4 otherBody = shared_mem[jBody];
      float3 rij = {otherBody.x - myBody.x, otherBody.y - myBody.y, otherBody.z - myBody.z};
      float rijSquared = rij.x * rij.x + rij.y * rij.y + rij.z * rij.z + softSquared;

      float ai = G * otherBody.w / (rijSquared * sqrtf(rijSquared));

      acc.x += ai * rij.x;
      acc.y += ai * rij.y;
      acc.z += ai * rij.z;
    }

    // store the result in global memory
    if(iBody < nBodies)
    {
      d_acc[iBody] = acc;
    }

  }

}


SimulationNBodyCudaAoS::SimulationNBodyCudaAoS(const unsigned long nBodies, const std::string &scheme, const float soft,
                                           const unsigned long randInit)
    : numBlocks((nBodies + THREADS_PER_BLK - 1) / THREADS_PER_BLK),
      SimulationNBodyInterface(nBodies, scheme, soft, randInit)
{
    this->flopsPerIte = 20.f * (float)this->getBodies().getN() * (float)this->getBodies().getN();
    this->accelerations.resize(this->getBodies().getN());

    //print CUDA device properties of the current device
    cuda::printGPUInfo();

    hipHostAlloc(&h_AoS_4, nBodies * sizeof(float4), hipHostMallocDefault);

    hipMalloc(&d_AoS, nBodies * sizeof(float4));
    hipMalloc(&d_acc, nBodies * sizeof(float3));

}

void SimulationNBodyCudaAoS::initIteration()
{
    for (unsigned long iBody = 0; iBody < this->getBodies().getN(); iBody++) {
        this->accelerations[iBody].ax = 0.f;
        this->accelerations[iBody].ay = 0.f;
        this->accelerations[iBody].az = 0.f;
    }
}

void SimulationNBodyCudaAoS::computeBodiesAcceleration()
{
    const std::vector<dataAoS_t<float>> &h_AoS_8 = this->getBodies().getDataAoS();
    const unsigned long n = this->getBodies().getN();

    #pragma omp parallel for 
    for(unsigned long i = 0; i < n; i++)
    {
      ((float4*)h_AoS_4)[i] = make_float4(h_AoS_8[i].qx, h_AoS_8[i].qy, h_AoS_8[i].qz, h_AoS_8[i].m);
    }

    //copy body data on device
    hipMemcpy(d_AoS, h_AoS_4, n * sizeof(float4), hipMemcpyHostToDevice);

    cuda::computeBodiesAccellAoS_k<<<numBlocks, THREADS_PER_BLK>>>((float4*)d_AoS, (float3*)d_acc, n, this->soft * this->soft, this->G);

    //copy back the result
    hipMemcpy(this->accelerations.data(), d_acc, n * sizeof(float3), hipMemcpyDeviceToHost);

}

void SimulationNBodyCudaAoS::computeOneIteration()
{
    this->initIteration();
    this->computeBodiesAcceleration();
    // time integration
    this->bodies.updatePositionsAndVelocities(this->accelerations, this->dt);
}

SimulationNBodyCudaAoS::~SimulationNBodyCudaAoS()
{
    //free memory
    hipFree(d_AoS);
    hipFree(d_acc);

    hipHostFree(h_AoS_4);
}
