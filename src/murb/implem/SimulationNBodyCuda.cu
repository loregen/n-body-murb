#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <fstream>
#include <iostream>
#include <limits>
#include <string>

#include "SimulationNBodyCuda.cuh"

#define MAX_SHARED_PER_BLOCK 48000
#define THREADS_PER_BLK 1024

namespace cuda
{
  void printGPUInfo()
  {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    printf("Number of devices: %d\n", nDevices);

    for (int i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (MHz): %d\n",
               prop.memoryClockRate / 1024);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
               2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        printf("  Total global memory (Gbytes) %.1f\n", (float)(prop.totalGlobalMem) / 1024.0 / 1024.0 / 1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n", (float)(prop.sharedMemPerBlock) / 1024.0);
        printf("  minor-major: %d-%d\n", prop.minor, prop.major);
        printf("  Warp-size: %d\n", prop.warpSize);
        printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n", prop.deviceOverlap ? "yes" : "no");
    }
  }

  __global__ void computeBodiesAccell_k(void *d_AoS, void *d_acc, const unsigned long nBodies, const float softSquared, const float G)
  {
    __shared__ float4 shared_mem[THREADS_PER_BLK];
    float4 *global_AoS = (float4 *)d_AoS;
    float3 *global_acc = (float3 *)d_acc;
    
    const unsigned long iBody = blockIdx.x * blockDim.x + threadIdx.x;
    
    float4 myBody;
    if(iBody < nBodies)
    {
      myBody = global_AoS[iBody];
    }
    float3 acc = {0.f, 0.f, 0.f};

    unsigned tileIdx;
    for(unsigned tile = 0; tile < nBodies / THREADS_PER_BLK; tile++)
    {
      tileIdx = tile * THREADS_PER_BLK + threadIdx.x;

      shared_mem[threadIdx.x] = global_AoS[tileIdx];
      __syncthreads();
      for(unsigned jBody = 0; jBody < THREADS_PER_BLK; jBody++)
      {
        float4 otherBody = shared_mem[jBody];
        float3 rij = {otherBody.x - myBody.x, otherBody.y - myBody.y, otherBody.z - myBody.z};
        float rijSquared = rij.x * rij.x + rij.y * rij.y + rij.z * rij.z + softSquared;

        float ai = G * otherBody.w / (rijSquared * sqrtf(rijSquared));

        acc.x += ai * rij.x;
        acc.y += ai * rij.y;
        acc.z += ai * rij.z;
      }
      __syncthreads();
    }

    tileIdx += THREADS_PER_BLK;
    // epilogue
    if(tileIdx < nBodies)
    {
      //load the last tile
      shared_mem[threadIdx.x] = global_AoS[tileIdx];
      __syncthreads();

      for(unsigned jBody = 0; jBody < nBodies % THREADS_PER_BLK; jBody++)
      {
        float4 otherBody = shared_mem[jBody];
        float3 rij = {otherBody.x - myBody.x, otherBody.y - myBody.y, otherBody.z - myBody.z};
        float rijSquared = rij.x * rij.x + rij.y * rij.y + rij.z * rij.z + softSquared;

        float ai = G * otherBody.w / (rijSquared * sqrtf(rijSquared));

        acc.x += ai * rij.x;
        acc.y += ai * rij.y;
        acc.z += ai * rij.z;
      }
    }

    if(iBody < nBodies)
    {
      global_acc[iBody] = acc;
    }

  }

}

SimulationNBodyCuda::SimulationNBodyCuda(const unsigned long nBodies, const std::string &scheme, const float soft,
                                           const unsigned long randInit)
    : SimulationNBodyInterface(nBodies, scheme, soft, randInit)
{
    this->flopsPerIte = 20.f * (float)this->getBodies().getN() * (float)this->getBodies().getN();
    this->accelerations.resize(this->getBodies().getN());

    //print CUDA device properties of the current device
    //cuda::printGPUInfo();

}

void SimulationNBodyCuda::initIteration()
{
    for (unsigned long iBody = 0; iBody < this->getBodies().getN(); iBody++) {
        this->accelerations[iBody].ax = 0.f;
        this->accelerations[iBody].ay = 0.f;
        this->accelerations[iBody].az = 0.f;
    }
}

void SimulationNBodyCuda::computeBodiesAcceleration()
{
    const std::vector<dataAoS_t<float>> &d = this->getBodies().getDataAoS();
    const unsigned long n = this->getBodies().getN();

    // device pointers
    void *d_AoS;
    void *d_acc;

    // allocate memory on the device
    hipMalloc(&d_AoS, 4 * n * sizeof(float));
    hipMalloc(&d_acc, 3 * n * sizeof(float));

    //copy body data on device
    hipMemcpy(d_AoS, d.data(), 4 * n * sizeof(float), hipMemcpyHostToDevice);

    int numBlocks = (n + THREADS_PER_BLK - 1) / THREADS_PER_BLK;

    cuda::computeBodiesAccell_k<<<numBlocks, THREADS_PER_BLK>>>(d_AoS, d_acc, n, this->soft * this->soft, this->G);

    //copy back the result
    hipMemcpy(this->accelerations.data(), d_acc, 3 * n * sizeof(float), hipMemcpyDeviceToHost);

    //free memory
    hipFree(d_AoS);
    hipFree(d_acc);
}

void SimulationNBodyCuda::computeOneIteration()
{
    this->initIteration();
    this->computeBodiesAcceleration();
    // time integration
    this->bodies.updatePositionsAndVelocities(this->accelerations, this->dt);
}
